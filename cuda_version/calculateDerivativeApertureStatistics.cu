#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono> //For time measurements


/**
 * @file calculateDerivativeApertureStatistics.cpp
 * This executable calculates the derivative of <MapMapMap> wrt to
 * the cosmological parameters \f$h$\f, \f$\sigma_8$\f, \f$\Omega_b$\f, 
 * \f$n_s$\f, \f$w$\f, \f$\Omega_m$\f, and \f$\Omega_\Lambda$\f 
 * for predefined thetas from the Takahashi+ Bispectrum
 * Uses 5-point stencil with stepsize h
 * Code uses CUDA and cubature library  (See https://github.com/stevengj/cubature for documentation)
 * @author Laila Linke
 * @warning thetas currently hardcoded
 * @warning Main cosmology is hardcoded (Set to either MS or SLICS), check constant slics in bispectrum.cuh for which cosmology is used! 
 * @todo Thetas should be read from command line
 * @todo cosmology should be read from command line
 */
int main(int argc, char* argv[])
{
      // Read in command line

  const char* message = R"( 
calculateDerivativeApertureStatistics.x : Wrong number of command line parameters (Needed: 6)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: 0: use three-point stencil, 1: use five-point stencil
Argument 4: Stencil stepsize
Argument 5: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 6 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateDerivativeApertureStatistics.x ../necessary_files/MR_cosmo.dat ../../results_MR/MapMapMap_derivatives.dat 1 0.01 1 ../necessary_files/nz_MR.dat
)";

  if(argc < 6)
    {
      std::cerr<<message<<std::endl;
      exit(1);
    };

  std::string cosmo_paramfile, outfn, nzfn;
  bool nz_from_file=false;
  bool five_point=false;
  double h;
  
  cosmo_paramfile=argv[1];
  outfn=argv[2];
  five_point=std::stoi(argv[3]);
  h=std::stod(argv[4]);
  nz_from_file=std::stoi(argv[5]);
  if(nz_from_file)
    {
      nzfn=argv[6];
    };

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);///<cosmology at which derivative is calculated

  
  double dz = cosmo.zmax/((double) n_redshift_bins); //redshift binsize
  std::vector<double> nz;
  if(nz_from_file)
    {
      // Read in n_z
      read_n_of_z(nzfn, dz, n_redshift_bins, nz);
    };
  
  // Check output file
  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };

  // User output
  std::cerr<<"Using cosmology from "<<cosmo_paramfile<<":"<<std::endl;
  std::cerr<<cosmo;
  std::cerr<<"Writing to:"<<outfn<<std::endl;
  if(five_point) std::cerr<<"Using five-point stencil"<<std::endl;
 


  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));

  // Set up thetas for which ApertureStatistics are calculated
  std::vector<double> thetas{0.5, 1, 2, 4, 8, 16, 32}; //Thetas in arcmin
  int N=thetas.size();

  
  // Borders of integral
  double phiMin=0.0;
  double phiMax=6.28319;
  double lMin=1;

  // Set up cosmologies at which Map^3 is calculated
  // This can probably be done smarter


  std::vector<cosmology> cosmos; ///<container for all cosmologies
  std::vector<double> derivative_parameters; //parameters the derivatives are taken in 

  cosmology newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.h=cosmo.h*(1.-2*h);
      cosmos.push_back(newCosmo);
    };
  newCosmo.h=cosmo.h*(1.-h);
  cosmos.push_back(newCosmo);
  newCosmo.h=cosmo.h*(1.+h);
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.h=cosmo.h*(1.+2*h);
      cosmos.push_back(newCosmo);
    };
    derivative_parameters.push_back(cosmo.h);
  
  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.sigma8=cosmo.sigma8*(1.-2*h);
      cosmos.push_back(newCosmo);
    };
  
  newCosmo.sigma8=cosmo.sigma8*(1.-h);
  cosmos.push_back(newCosmo);
  newCosmo.sigma8=cosmo.sigma8*(1.+h);
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.sigma8=cosmo.sigma8*(1.+2*h);
      cosmos.push_back(newCosmo);
    };
    derivative_parameters.push_back(cosmo.sigma8);
  
  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.omb=cosmo.omb*(1-2*h);
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.push_back(newCosmo);
    };
  
  newCosmo.omb=cosmo.omb*(1-h);
  newCosmo.omc=newCosmo.om-newCosmo.omb;
  cosmos.push_back(newCosmo);
  newCosmo.omb=cosmo.omb*(1+h);
  newCosmo.omc=newCosmo.om-newCosmo.omb;
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.omb=cosmo.omb*(1+2*h);
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.push_back(newCosmo);
    };
    derivative_parameters.push_back(cosmo.omb);
  
  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.ns=cosmo.ns*(1-2*h);
      cosmos.push_back(newCosmo);
    };
  newCosmo.ns=cosmo.ns*(1-h);
  cosmos.push_back(newCosmo);
  newCosmo.ns=cosmo.ns*(1+h);
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.ns=cosmo.ns*(1+2*h);
      cosmos.push_back(newCosmo);
    };
  derivative_parameters.push_back(cosmo.ns);

  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.w=cosmo.w*(1-2*h);
      cosmos.push_back(newCosmo);
    };
  newCosmo.w=cosmo.w*(1-h);
  cosmos.push_back(newCosmo);
  newCosmo.w=cosmo.w*(1+h);
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.w=cosmo.w*(1+2*h);
      cosmos.push_back(newCosmo);
    };
  derivative_parameters.push_back(cosmo.w);

  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.om=cosmo.om*(1-2*h);
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.push_back(newCosmo);
    };
  newCosmo.om=cosmo.om*(1-h);
  newCosmo.omc=newCosmo.om-newCosmo.omb;
  cosmos.push_back(newCosmo);
  newCosmo.om=cosmo.om*(1+h);
  newCosmo.omc=newCosmo.om-newCosmo.omb;
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.om=cosmo.om*(1+2*h);
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.push_back(newCosmo);
    };
    derivative_parameters.push_back(cosmo.om);

  newCosmo=cosmo;
  if(five_point)
    {
      newCosmo.ow=cosmo.ow*(1-2*h);
      cosmos.push_back(newCosmo);
    };
				  
  newCosmo.ow=cosmo.ow*(1-h);
  cosmos.push_back(newCosmo);
  newCosmo.ow=cosmo.ow*(1+h);
  cosmos.push_back(newCosmo);
  if(five_point)
    {
      newCosmo.ow=cosmo.ow*(1+2*h);
      cosmos.push_back(newCosmo);
    };
  derivative_parameters.push_back(cosmo.ow);

  int Ncosmos=cosmos.size();///<Number of cosmologies

  // Calculation of Map^3
  double MapMapMaps[Ncosmos][N*N*N]; ///<Array which will contain MapMapMap calculated

  
  
  for(int i=0; i<Ncosmos; i++)
    {
      std::cout<<"Doing calculations for cosmology "<<i<<" of "<<Ncosmos<<std::endl;
      auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement
      // Initialize Bispectrum
      if(nz_from_file)
	{
	  set_cosmology(cosmos[i], dz, nz_from_file, &nz);
	}
      else
	{
	  set_cosmology(cosmos[i], dz);
	};


      //Needed for monitoring
      int Ntotal=N*(N+1)*(N+2)/6.; //Total number of bins that need to be calculated, = (N+3-1) ncr 3
      int step=0;

      //Calculate <MapMapMap>(theta1, theta2, theta3) in three loops
      // Calculation only for theta1<=theta2<=theta3, other combinations are assigned
      for (int j=0; j<N; j++)
	{
	  double theta1=thetas.at(j)*3.1416/180./60; //Conversion to rad
	  for (int k=j; k<N; k++)
	    {
	      double theta2=thetas.at(k)*3.1416/180./60.;
	      for(int l=k; l<N; l++)
		{
		  double theta3=thetas.at(l)*3.1416/180./60.;
		  double thetas_calc[3]={theta1, theta2, theta3};
		  //Progress for the impatient user (Thetas in arcmin)
		  step+=1;
		  std::cout<<step<<"/"<<Ntotal<<": Thetas:"<<thetas.at(j)<<" "<<thetas.at(k)<<" "<<thetas.at(l)<<" \r"; //\r is so that only one line is shown
		  std::cout.flush();

		  double Map3=MapMapMap(thetas_calc, phiMin, phiMax, lMin); //Do calculation
	      
		  // Do assigment (including permutations)
		  MapMapMaps[i][j*N*N+k*N+l]=Map3;
		  MapMapMaps[i][j*N*N+l*N+k]=Map3;
		  MapMapMaps[i][k*N*N+j*N+l]=Map3;
		  MapMapMaps[i][k*N*N+l*N+j]=Map3;
		  MapMapMaps[i][l*N*N+j*N+k]=Map3;
		  MapMapMaps[i][l*N*N+k*N+j]=Map3;
		};
	    };
	};

      // Stop measuring time and calculate the elapsed time
      auto end = std::chrono::high_resolution_clock::now();
      auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
  
      std::cout<<"Time needed for last cosmology:"<<elapsed.count()*1e-9<<std::endl;
    };
    
  
  // Calculation of Derivatives
  int Nderivs=int(Ncosmos/2);
  if(five_point) Nderivs=int(Ncosmos/4); ///<Number of derivatives
  double derivs_MapMapMaps[Nderivs][N*N*N]; ///<Array which will contain MapMapMap calculated

#pragma omp parallel for collapse(2)
  for(int i=0; i<Nderivs; i++)
    {
      for(int j=0; j<N*N*N; j++)
	{
	  if(five_point)
	    {
	      // Stencil calculation: df/dx = [f(x-2h)-8f(x-h)+8f(x+h)-f(x+2h)]/(12h)
	      derivs_MapMapMaps[i][j]=(MapMapMaps[4*i][j]-8*MapMapMaps[4*i+1][j]+8*MapMapMaps[4*i+2][j]-MapMapMaps[4*i+3][j])/(12.*h*derivative_parameters.at(i));
	    }
	  else
	    {
	      derivs_MapMapMaps[i][j]=(MapMapMaps[2*i+1][j]-MapMapMaps[2*i][j])/2/h/derivative_parameters.at(i);
	    }
	}
    }


  // Output (Cannot be parallelized!!)
  std::cout<<"Writing results to "<<outfn<<std::endl;
  for(int i=0; i<Nderivs; i++)
    {
      for(int j=0; j<N*N*N; j++)
	{
	  out<<derivs_MapMapMaps[i][j]<<" ";
	}
      out<<std::endl;
    }

      
}
