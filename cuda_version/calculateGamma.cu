#include "hip/hip_runtime.h"
#include "gamma.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"
#include "cosmology.cuh"


#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

#define slics false


int main(int argc, char** argv)
{
  // Read in command line
  const char* message = R"( 
calculateGamma.x : Wrong number of command line parameters (Needed: 4)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Config file for the 3pcf
Argument 3: Outputfilename, directory needs to exist 
Argument 4: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 5 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)
Argument 6 (optional): GPU device number
Example:
./calculateGamma.x ../necessary_files/MR_cosmo.dat ../../results_MR/MapMapMap_varyingCosmos.dat 1 ../necessary_files/nz_MR.dat
)";

  if(argc < 4)
    {
      std::cerr<<message<<std::endl;
      exit(1);
    };

  std::string cosmo_paramfile, outfn, nzfn, config_file;
  bool nz_from_file=false;

  cosmo_paramfile=argv[1];
  config_file = argv[2];
  outfn=argv[3];
  nz_from_file=std::stoi(argv[4]);
  if(nz_from_file)
    {
      nzfn=argv[5];
    };

  
  std::cout << "Executing " << argv[0] << " ";
  if(argc==7)
    {
      int deviceNumber = atoi(argv[6]);
      std::cout << "on GPU " << deviceNumber << std::endl;
      hipSetDevice(deviceNumber);
    }
  else
    {
      std::cout << "on default GPU" << std::endl;
    };
  

  
  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);
  double dz = cosmo.zmax / ((double) n_redshift_bins);

  std::vector<double> nz;
  if(nz_from_file)
    {
      // Read in n_z
      read_n_of_z(nzfn, dz, n_redshift_bins, nz);
    };
  
  // Check output file
  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };

  // User output
  std::cerr<<"Using cosmology:"<<std::endl;
  std::cerr<<cosmo;
  std::cerr<<"Writing to:"<<outfn<<std::endl;

  configGamma config;
  read_gamma_config(config_file,config);
  std::cerr<<config;
  // Binning
  int steps = config.rsteps;
  int usteps = config.usteps;
  int vsteps = config.vsteps;

  double rmin = config.rmin;
  double rmax = config.rmax;
  double umin = config.umin;
  double umax = config.umax;
  double vmin = config.vmin;
  double vmax = config.vmax;

  

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));


  if(nz_from_file)
    {
      set_cosmology(cosmo, dz, nz_from_file, &nz);
    }
  else
    {
      set_cosmology(cosmo, dz);
    };

  compute_weights_bessel();

  //Calculation + Output in one
  double lrmin = log(rmin);
  double lrmax = log(rmax);
  
  auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement

  
  for(int i=0; i<steps; i++)
    {
      double r=exp(lrmin+(lrmax-lrmin)/steps*(i+0.5));
      for(int j=0; j<usteps; j++)
	{
	  double u = umin+(umax-umin)/usteps*(j+0.5);
    // double u;
    // if(j<10) u = 0.2/10*(j+0.5);
    // else u = 1./10*((j-8)+0.5);

	  for(int k=0; k<vsteps; k++)
	    {
	      double v= vmin+(vmax-vmin)/vsteps*(k+0.5);

	      double r2 = r*M_PI/180./60.; //THIS IS THE BINNING BY JARVIS. FROM THE WEBSITE, NOT THE PAPER.
	      double r3 = r2*u;
	      double r1 = v*r3+r2;
	      
	      std::complex<double> _gamma0 = gamma0(r1, r2, r3,z_max);
	      std::complex<double> _gamma1 = gamma1(r1, r2, r3,z_max);
	      std::complex<double> _gamma2 = gamma2(r1, r2, r3,z_max);
	      std::complex<double> _gamma3 = gamma3(r1, r2, r3,z_max);
	      auto end = std::chrono::high_resolution_clock::now();
	      auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
        int completed_steps = i*usteps*vsteps+j*vsteps+k+1;
        int total_steps = steps*usteps*vsteps;
        double progress = (completed_steps*1.)/(total_steps);

	      printf("\r [%3d%%] in %.2f h. Est. remaining: %.2f h. Average: %.2f s per step.",
        static_cast<int>(progress*100),
        elapsed.count()*1e-9/3600,
        (total_steps-completed_steps)*elapsed.count()*1e-9/3600/completed_steps,
        elapsed.count()*1e-9/completed_steps);
	      out
        <<i<<" "
        <<j<<" "
        <<k<<" "
        <<real(_gamma0)<<" "
        <<imag(_gamma0)<<" "
        <<real(_gamma1)<<" "
        <<imag(_gamma1)<<" "
        <<real(_gamma2)<<" "
        <<imag(_gamma2)<<" "
        <<real(_gamma3)<<" "
        <<imag(_gamma3)<<" "
        <<r<<" "
        <<u<<" "
        <<v<<     
        std::endl;
	    };
	};
    };


  
  
}
