#include "hip/hip_runtime.h"
#include "gamma.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"


#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

#define slics false


int main(int argc, char** argv)
{
    std::cout << "Executing " << argv[0] << " ";
    if(argc>=2)
    {
        int deviceNumber = atoi(argv[1]);
        std::cout << "on GPU " << deviceNumber << std::endl;
        hipSetDevice(deviceNumber);
    }
    else
    {
        std::cout << "on default GPU";
    }
  // Set Up Cosmology
  struct cosmology cosmo;

  if(slics)
    {
      printf("using SLICS cosmology...\n");
      cosmo.h=0.6898;     // Hubble parameter
      cosmo.sigma8=0.826; // sigma 8
      cosmo.omb=0.0473;   // Omega baryon
      cosmo.omc=0.2905-cosmo.omb;   // Omega CDM
      cosmo.ns=0.969;    // spectral index of linear P(k)
      cosmo.w=-1.0;
      cosmo.om = cosmo.omb+cosmo.omc;
      cosmo.ow = 1-cosmo.om;
    }
    else
    {
      printf("using Millennium cosmology...\n");
      cosmo.h = 0.73;
      cosmo.sigma8 = 0.9;
      cosmo.omb = 0.045;
      cosmo.omc = 0.25 - cosmo.omb;
      cosmo.ns = 1.;
      cosmo.w = -1.0;
      cosmo.om = cosmo.omc+cosmo.omb;
      cosmo.ow = 1.-cosmo.om;
    };

  // Binning
  int steps = 10;
  int usteps = 10;
  int vsteps = 10;

  double rmin = 0.1;
  double rmax = 120.;
  double umin = 0;
  double umax = 1;
  double vmin = 0;
  double vmax = 1;

    // Set output file
  // std::string outfn="Gammas_"+std::to_string(rmin)+"_to_"+std::to_string(rmax)+".dat";
  if(argc<4)
  {
      std::cout << "need to provide parameter index (int) and stencil (float)!";
      exit(1);
  }
  int parameter_index = atoi(argv[2]);
  double stencil = atof(argv[3]);
  std::string outfn;
  if(parameter_index == 0)
  {
    std::cout << "modifying Omega_m by " << stencil << std::endl;
    cosmo.om *= (1+stencil);
    cosmo.ow = 1.-cosmo.om;
    cosmo.omc = cosmo.om - cosmo.omb;
    std::cout << "New Omega_m: " << cosmo.om << std::endl;


    outfn="/vol/euclid6/euclid6_ssd/sven/threepoint_with_laila/results_MR/fisher/Gammas_0p1_to_120_Omega_m_"+std::to_string(stencil)+".dat";
  }
  else if(parameter_index==1)
  {
    std::cout << "modifying sigma_8 by " << stencil << std::endl;
    cosmo.sigma8 *= (1+stencil);
    std::cout << "New sigma_8: " << cosmo.sigma8 << std::endl;

    outfn="/vol/euclid6/euclid6_ssd/sven/threepoint_with_laila/results_MR/fisher/Gammas_0p1_to_120_sigma_8_"+std::to_string(stencil)+".dat";
  }
  else
  {
      std::cout << "Invalid parameter index!" << std::endl;
      exit(1);
  }

  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };


  if(slics) cosmo.zmax = 3.;
  else cosmo.zmax = 1.1;
  double dz = cosmo.zmax / ((double) n_redshift_bins);

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));

  set_cosmology(cosmo, dz);

  compute_weights_bessel();

  //Calculation + Output in one
  double lrmin = log(rmin);
  double lrmax = log(rmax);
  
  auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement

  
  for(int i=0; i<steps; i++)
    {
      double r=exp(lrmin+(lrmax-lrmin)/steps*(i+0.5));
      for(int j=0; j<usteps; j++)
	{
	  double u = umin+(umax-umin)/usteps*(j+0.5);
    // double u;
    // if(j<10) u = 0.2/10*(j+0.5);
    // else u = 1./10*((j-8)+0.5);

	  for(int k=0; k<vsteps; k++)
	    {
	      double v= vmin+(vmax-vmin)/vsteps*(k+0.5);

	      double r2 = r*M_PI/180./60.; //THIS IS THE BINNING BY JARVIS. FROM THE WEBSITE, NOT THE PAPER.
	      double r3 = r2*u;
	      double r1 = v*r3+r2;
	      
	      std::complex<double> _gamma0 = gamma0(r1, r2, r3,z_max);
	      std::complex<double> _gamma1 = gamma1(r1, r2, r3,z_max);
	      std::complex<double> _gamma2 = gamma2(r1, r2, r3,z_max);
	      std::complex<double> _gamma3 = gamma3(r1, r2, r3,z_max);
	      auto end = std::chrono::high_resolution_clock::now();
	      auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
        int completed_steps = i*usteps*vsteps+j*vsteps+k+1;
        int total_steps = steps*usteps*vsteps;
        double progress = (completed_steps*1.)/(total_steps);

	      printf("\r [%3d%%] in %.2f h. Est. remaining: %.2f h. Average: %.2f s per step.",
        static_cast<int>(progress*100),
        elapsed.count()*1e-9/3600,
        (total_steps-completed_steps)*elapsed.count()*1e-9/3600/completed_steps,
        elapsed.count()*1e-9/completed_steps);
	      out
        <<i<<" "
        <<j<<" "
        <<k<<" "
        <<real(_gamma0)<<" "
        <<imag(_gamma0)<<" "
        <<real(_gamma1)<<" "
        <<imag(_gamma1)<<" "
        <<real(_gamma2)<<" "
        <<imag(_gamma2)<<" "
        <<real(_gamma3)<<" "
        <<imag(_gamma3)<<" "
        <<r<<" "
        <<u<<" "
        <<v<<     
        std::endl;
	    };
	};
    };


  
  
}
