#include "hip/hip_runtime.h"
#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"
#include "helpers.cuh"
#include "halomodel.cuh"

#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <math.h>

/**
 * @file calculateTrispectrum_halomodel.cu
 * This executable gives out the 2D- Trispectrum based on the 1-halo term of the halomodel
 * The cosmology is read from file
 * @warning ellMin and ellMax are hardcoded
 * @author Laila Linke
 */
int main(int argc, char *argv[])
{
  // Read in command line

  const char *message = R"( 
calculateTrispectrum_halomodel.x : Wrong number of command line parameters (Needed: 4)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateTrispectrum_halomodel.x ../necessary_files/MR_cosmo.dat  ../../results_MR/Trispectrum.dat ../necessary_files/nz_MR.dat
)";

  if (argc < 4) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, thetasfn, outfn, nzfn;

  cosmo_paramfile = argv[1];
  outfn = argv[2];
  nzfn = argv[3];

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  // Read in n_z
  std::vector<double> nz;
  if (nz_from_file)
  {
    read_n_of_z(nzfn, n_redshift_bins, cosmo.zmax, nz);
  };

  // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open())
  {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };

  // User output
  std::cerr << "Using cosmology from " << cosmo_paramfile << ":" << std::endl;
  std::cerr << cosmo;
  std::cerr << "Writing to:" << outfn << std::endl;

  // Initialize Bispectrum

  copyConstants();

  if (nz_from_file)
  {
    std::cerr << "Using n(z) from " << nzfn << std::endl;
    set_cosmology(cosmo, &nz);
  }
  else
  {
    set_cosmology(cosmo);
  };

  initHalomodel();

  double lmin = log10(150);
  double lmax = log10(40000);
  int Nbins = 5; // 13;//100;
  double lbin = (lmax - lmin) / Nbins;

  for (int i = 0; i < Nbins; i++)
  {
    double l1 = pow(10, lmin + i * lbin);
    //    double l=pow(10, lmin+(i+0.5)*lbin);
    for (int j = 0; j < Nbins; j++)
    {
      double l2 = pow(10, lmin + j * lbin);
      for (int k = 0; k < Nbins; k++)
      {
        double l3 = pow(10, lmin + k * lbin);
        for (int l = 0; l < Nbins; l++)
        {
          double l4 = pow(10, lmin + l * lbin);
          double T = Trispectrum(l1, l2, l3, l4);
          out << l1 << " "
              << l2 << " "
              << l3 << " "
              << l4 << " "
              << T << std::endl;
        }
      }
    };
  }

  return 0;
}
