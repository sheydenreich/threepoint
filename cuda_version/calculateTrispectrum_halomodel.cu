#include "hip/hip_runtime.h"
#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"
#include "helpers.cuh"
#include "halomodel.cuh"

#include <fstream>
#include <iostream>
#include <string>
#include <vector>
/**
 * @file calculateApertureStatistics.cu
 * This executable calculates <MapMapMap> from the
 * Takahashi+ Bispectrum
 * Aperture radii are read from file and <MapMapMap> is only calculated for
 * independent combis of thetas Code uses CUDA and cubature library  (See
 * https://github.com/stevengj/cubature for documentation)
 * @author Laila Linke
 */
int main(int argc, char *argv[]) {
  // Read in command line

  const char *message = R"( 
calculateApertureStatistics.x : Wrong number of command line parameters (Needed: 5)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Filename with thetas [arcmin]
Argument 3: Outputfilename, directory needs to exist 
Argument 4: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 5 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateApertureStatistics.x ../necessary_files/MR_cosmo.dat ../necessary_files/HOWLS_thetas.dat ../../results_MR/MapMapMap_bispec_gpu_nz.dat 1 ../necessary_files/nz_MR.dat
)";

  if (argc < 5) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, thetasfn, outfn, nzfn;
  bool nz_from_file = false;

  cosmo_paramfile = argv[1];
  outfn = argv[2];
  nz_from_file = std::stoi(argv[3]);
  if (nz_from_file) {
    nzfn = argv[4];
  };

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  // Read in n_z
  std::vector<double> nz;
  if (nz_from_file) {
    read_n_of_z(nzfn, n_redshift_bins, cosmo.zmax, nz);
  };

  // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open()) {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };


  // User output
  std::cerr << "Using cosmology from " << cosmo_paramfile << ":" << std::endl;
  std::cerr << cosmo;
  std::cerr << "Writing to:" << outfn << std::endl;

  // Initialize Bispectrum

  copyConstants();

  if (nz_from_file) {
    std::cerr << "Using n(z) from " << nzfn << std::endl;
    set_cosmology(cosmo, &nz);
  } 
  else 
  {
    set_cosmology(cosmo);
  };

  initHalomodel();
  
  double lmin=log10(150);
  double lmax=log10(40000);
  int Nbins=13;//100;
  double lbin=(lmax-lmin)/Nbins;

  for( int i=0; i<Nbins; i++)
  {
//    double l=pow(10, lmin+(i+0.5)*lbin);
    double l=pow(10, lmin+i*lbin);
    double T=Trispectrum(l,l,l,l);
    out<<l<<" "<<T<<std::endl;
  }


  return 0;
}
