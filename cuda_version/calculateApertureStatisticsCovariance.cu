#include "hip/hip_runtime.h"
#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"

#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <chrono>

/**
 * @file calculateApertureStatistics.cu
 * This executable calculates <MapMapMap> from the
 * Takahashi+ Bispectrum
 * Aperture radii are read from file and <MapMapMap> is only calculated for
 * independent combis of thetas Code uses CUDA and cubature library  (See
 * https://github.com/stevengj/cubature for documentation)
 * @author Sven Heydenreich
 */
int main(int argc, char *argv[]) {
  // Read in command line

  const char *message = R"( 
calculateApertureStatistics.x : Wrong number of command line parameters (Needed: 5)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Filename for covariance parameters (ASCII, see necessary_files/HOWLS_covpar.dat for an example)
Argument 3: Filename with thetas [arcmin]
Argument 4: Outputfilename, directory needs to exist 
Argument 5: 0: calculate only variance, or 1: calculate full covariance
Argument 6: Shapenoise, 0: ignore, 1: calculate
Argument 7: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 8 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateApertureStatisticsCovariance.x ../necessary_files/SLICS_cosmo.dat ../necessary_files/HOWLS_covariance.dat ../necessary_files/HOWLS_thetas.dat ../../results_MR/MapMapMap_covariance.dat 1 1 1 ../necessary_files/nz_MR.dat
)";

  if (argc < 7) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, covariance_paramfile, thetasfn, outfn, nzfn;
  bool nz_from_file = false;
  bool calculate_covariance = false;
  bool shapenoise = false;

  cosmo_paramfile = argv[1];
  covariance_paramfile = argv[2];
  thetasfn = argv[3];
  outfn = argv[4];
  calculate_covariance = std::stoi(argv[5]);
  shapenoise = std::stoi(argv[6]);
  nz_from_file = std::stoi(argv[7]);
  if (nz_from_file) {
    nzfn = argv[8];
  };

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);
  double dz = cosmo.zmax / ((double)n_redshift_bins - 1); // redshift binsize

  // Read in n_z
  std::vector<double> nz;
  if (nz_from_file) {

    read_n_of_z(nzfn, dz, n_redshift_bins, nz);
  };

  // Read in covariance parameters
  covarianceParameters covPar;
  read_covariance_param(covariance_paramfile, covPar);


  // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open()) {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };

  // Read in thetas
  std::vector<double> thetas;
  read_thetas(thetasfn, thetas);
  int N = thetas.size();

  // User output
  std::cerr << "Using cosmology from " << cosmo_paramfile << ":" << std::endl;
  std::cerr << cosmo;
  std::cerr << "Using thetas in " << thetasfn << std::endl;
  std::cerr << "Covariance from " << covariance_paramfile << ":" << std::endl;
  std::cerr << covPar;
  if(shapenoise)
    std::cerr << "Calculating covariance WITH shapenoise" << std::endl;
  else
    std::cerr << "Calculating covariance WITHOUT shapenoise" << std::endl;
  std::cerr << "Writing to:" << outfn << std::endl;

  // Initialize Bispectrum

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96), &A96, 48 * sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96), &W96, 48 * sizeof(double)));

  if (nz_from_file) {
    std::cerr << "Using n(z) from " << nzfn << std::endl;
    set_cosmology(cosmo, dz, nz_from_file, &nz);
  } else {
    set_cosmology(cosmo, dz);
  };

//   // Borders of integral
//   double phiMin = 0.0;
//   double phiMax = 6.28319;
//   double lMin = 1;

  // Set up vector for aperture statistics

int completed_steps = 0;
int Ntotal;
std::vector<double> Cov_MapMapMaps;
  if(calculate_covariance)
  {
    Ntotal = pow(N*(N+1)*(N+2)/6,2);
  }
  else
  {
    Ntotal = N*(N+1)*(N+2)/6;
  }
  Cov_MapMapMaps.reserve(Ntotal);

auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement
//Calculate <MapMapMap>(theta1, theta2, theta3) 
//This does the calculation only for theta1<=theta2<=theta3, but because of
//the properties of omp collapse, the for-loops are defined starting from 0
for (int i=0; i<N; i++)
    {
    double theta1=thetas.at(i)*3.1416/180./60; //Conversion to rad
    for (int j=i; j<N; j++)
    {
    double theta2=thetas.at(j)*3.1416/180./60.;
    for(int k=j; k<N; k++)
        {
        double theta3=thetas.at(k)*3.1416/180./60.;
        double thetas_123[3]={theta1, theta2, theta3};
        if(!calculate_covariance)
                {
        double thetas_456[3]={theta1, theta2, theta3};
        
        double MapMapMap=Gaussian_MapMapMap_Covariance(thetas_123,thetas_456,covPar,shapenoise); //Do calculation
            
            // Do assigment (including permutations)
            Cov_MapMapMaps.push_back(MapMapMap);
            auto end = std::chrono::high_resolution_clock::now();
            auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
            completed_steps++;
            double progress = (completed_steps*1.)/(Ntotal);
            
            printf("\r [%3d%%] in %.2f h. Est. remaining: %.2f h. Average: %.2f s per step.",
               static_cast<int>(progress*100),
               elapsed.count()*1e-9/3600,
               (Ntotal-completed_steps)*elapsed.count()*1e-9/3600/completed_steps,
               elapsed.count()*1e-9/completed_steps);
            
                  }
            else
                  {
            for(int ii=0; ii<N; ii++)
                      {
                double theta4=thetas.at(ii)*3.1416/180./60; //Conversion to rad
                for(int jj=ii; jj<N; jj++)
                          {
                double theta5=thetas.at(jj)*3.1416/180./60.;
                for(int kk=jj; kk<N; kk++)
                              {                          
                                
                    double theta6=thetas.at(kk)*3.1416/180./60.;
                    double thetas_456[3]={theta4, theta5, theta6};
  
                    double MapMapMap=Gaussian_MapMapMap_Covariance(thetas_123,thetas_456,covPar,shapenoise); //Do calculation

                    // Do assigment (including permutations)
                    // int index_123[3] = {i,j,k};
                    // int index_456[3] = {ii,jj,kk};
  
                    // std::sort(index_123,index_123+3);
                    // std::sort(index_456,index_456+3);
                    // do{
                //   do{
                    Cov_MapMapMaps.push_back(MapMapMap);
                //   }
                //   while(std::next_permutation(index_123,index_123+3));
                    // }
                    // while(std::next_permutation(index_456,index_456+3));
                    auto end = std::chrono::high_resolution_clock::now();
                    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
                    completed_steps++;
                    double progress = (completed_steps*1.)/(Ntotal);
                    
                    printf("\r [%3d%%] in %.2f h. Est. remaining: %.2f h. Average: %.2f s per step. Current thetas: (%.1f, %.1f, %.1f, %.1f, %.1f, %.1f)",
                       static_cast<int>(progress*100),
                       elapsed.count()*1e-9/3600,
                       (Ntotal-completed_steps)*elapsed.count()*1e-9/3600/completed_steps,
                       elapsed.count()*1e-9/completed_steps,
                       theta1*180*60/3.1416, theta2*180*60/3.1416, theta3*180*60/3.1416, theta4*180*60/3.1416, theta5*180*60/3.1416, theta6*180*60/3.1416);
                              }
                          }
                      }
                  }
            
              };
          };
      };
  std::cout << std::endl << "Done! Writing output..." << std::endl;

  // Output
  //Print out ==> Should not be parallelized!!!
  int steps = 0;
  if(!calculate_covariance)
    {
    for (int i=0; i<N; i++)
      {
        for(int j=i; j<N; j++)
	  {
            for(int k=j; k<N; k++)
	      {
		out<<thetas[i]<<" "
		   <<thetas[j]<<" "
		   <<thetas[k]<<" "
		   <<Cov_MapMapMaps.at(steps)<<" "
           <<std::endl;
           steps++;
	      };
	  };
      };
    }
  else
    {
      for (int i=0; i<N; i++)
	{
	  for(int j=i; j<N; j++)
	    {
	      for(int k=j; k<N; k++)
		{
		  for(int ii=0; ii<N; ii++)
		    {
		      for(int jj=ii; jj<N; jj++)
			{
			  for(int kk=jj; kk<N; kk++)
			    {
			      out<<thetas[i]<<" "
				 <<thetas[j]<<" "
				 <<thetas[k]<<" "
				 <<thetas[ii]<<" "
				 <<thetas[jj]<<" "
				 <<thetas[kk]<<" "
				 <<Cov_MapMapMaps.at(steps)<<" "
                 <<std::endl;
                steps++;
			    }
			}
		    }
		};
	    };
	};
      
    }
std::cout << "Done." << std::endl;
    

  return 0;
}
