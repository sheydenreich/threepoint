#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"
#include "cosmology.cuh"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
/**
 * @file calculateApertureStatistics.cu
 * This executable calculates <MapMapMap> for predefined thetas from the
 * Takahashi+ Bispectrum
 * Code uses CUDA and cubature library  (See https://github.com/stevengj/cubature for documentation)
 * @author Laila Linke
 * @warning thetas currently hardcoded
 * @warning Output is hardcoded
 * @todo Thetas should be read from command line
 * @todo Outputfilename should be read from command line
 */
int main(int argc, char* argv[])
{
  // Read in command line

  const char* message = R"( 
calculateApertureStatistics.x : Wrong number of command line parameters (Needed: 4)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 4 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateApertureStatistics.x ../necessary_files/MR_cosmo.dat ../../results_MR/MapMapMap_bispec_gpu_nz.dat 1 ../necessary_files/nz_MR.dat
)";

  if(argc < 4)
    {
      std::cerr<<message<<std::endl;
      exit(1);
    };

  std::string cosmo_paramfile, outfn, nzfn;
  bool nz_from_file=false;

  cosmo_paramfile=argv[1];
  outfn=argv[2];
  nz_from_file=std::stoi(argv[3]);
  if(nz_from_file)
    {
      nzfn=argv[4];
    };
  
 
  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);
  double dz = cosmo.zmax/((double) n_redshift_bins); //redshift binsize

  std::vector<double> nz;
  if(nz_from_file)
    {
      // Read in n_z
      read_n_of_z(nzfn, dz, n_redshift_bins, nz);
    };
  
  // Check output file
  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };

  // User output
  std::cerr<<"Using cosmology from "<<cosmo_paramfile<<":"<<std::endl;
  std::cerr<<cosmo;
  std::cerr<<"Writing to:"<<outfn<<std::endl;
  
  //Initialize Bispectrum

 
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));

  if(nz_from_file)
    {
      set_cosmology(cosmo, dz, nz_from_file, &nz);
    }
  else
    {
      set_cosmology(cosmo, dz);
    };
  
  // Set up thetas for which ApertureStatistics are calculated
  std::vector<double> thetas{0.5, 1, 2, 4, 8, 16, 32}; //Thetas in arcmin
  int N=thetas.size();

  // Borders of integral
  double phiMin=0.0;
  double phiMax=6.28319;
  double lMin=1;
  
  // Set up vector for aperture statistics
  std::vector<double> MapMapMaps(N*N*N);

  //Needed for monitoring
  int Ntotal=N*(N+1)*(N+2)/6.; //Total number of bins that need to be calculated, = (N+3+1) ncr 3
  int step=0;

  //Calculate <MapMapMap>(theta1, theta2, theta3) in three loops
  // Calculation only for theta1<=theta2<=theta3, other combinations are assigned
  for (int i=0; i<N; i++)
    {
      double theta1=thetas.at(i)*3.1416/180./60; //Conversion to rad
      
      for (int j=i; j<N; j++)
	{
	  double theta2=thetas.at(j)*3.1416/180./60.;

	  for(int k=j; k<N; k++)
	    {

	      double theta3=thetas.at(k)*3.1416/180./60.;
	      double thetas_calc[3]={theta1, theta2, theta3};
	      //Progress for the impatient user (Thetas in arcmin)
	      step+=1;
	      std::cout<<step<<"/"<<Ntotal<<": Thetas:"<<thetas.at(i)<<" "<<thetas.at(j)<<" "<<thetas.at(k)<<" \r";
	      std::cout.flush();

	      double Map3=MapMapMap(thetas_calc, phiMin, phiMax, lMin); //Do calculation
	      
	      // Do assigment (including permutations)
	      MapMapMaps.at(i*N*N+j*N+k)=Map3;
	      MapMapMaps.at(i*N*N+k*N+j)=Map3;
	      MapMapMaps.at(j*N*N+i*N+k)=Map3;
	      MapMapMaps.at(j*N*N+k*N+i)=Map3;
	      MapMapMaps.at(k*N*N+i*N+j)=Map3;
	      MapMapMaps.at(k*N*N+j*N+i)=Map3;
	    };
	};
    };

  //Output
  for (int i=0; i<N; i++)
    {
      for(int j=0; j<N; j++)
	{
	  for(int k=0; k<N; k++)
	    {
	      out<<thetas[i]<<" "
		 <<thetas[j]<<" "
		 <<thetas[k]<<" "
		 <<MapMapMaps.at(k*N*N+i*N+j)<<" "
		 <<std::endl;
	    };
	};
    };
    




  
  return 0;
}
