#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono> //For time measurements
/**
 * @file calculateApertureStatistics_varyingCosmos.cu
 * This executable calculates <MapMapMap> for variations of
 * the cosmological parameters \f$h$\f, \f$\sigma_8$\f, \f$\Omega_b$\f, 
 * \f$n_s$\f, \f$w$\f, \f$\Omega_m$\f, and \f$\Omega_\Lambda$\f 
 * for predefined thetas from the Takahashi+ Bispectrum
 * Code uses CUDA and cubature library  (See https://github.com/stevengj/cubature for documentation)
 * @author Laila Linke
 * @warning Currently only equilateral triangles
 * @warning thetas currently hardcoded
 * @warning Main cosmology is hardcoded (Set to either MS)
 * @todo Thetas should be read from command line
 * @todo cosmology should be read from command line
 */

int main(int argc, char* argv[])
{

    // Read in command line

  const char* message = R"( 
calculateApertureStatistics_varyingCosmos.x : Wrong number of command line parameters (Needed: 4)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 4 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateApertureStatistics.x ../necessary_files/MR_cosmo.dat ../../results_MR/MapMapMap_varyingCosmos.dat 1 ../necessary_files/nz_MR.dat
)";

  if(argc < 4)
    {
      std::cerr<<message<<std::endl;
      exit(1);
    };

  std::string cosmo_paramfile, outfn, nzfn;
  bool nz_from_file=false;

  cosmo_paramfile=argv[1];
  outfn=argv[2];
  nz_from_file=std::stoi(argv[3]);
  if(nz_from_file)
    {
      nzfn=argv[4];
    };
  
  
  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  double dz = cosmo.zmax/((double) n_redshift_bins); //redshift binsize
  
  std::vector<double> nz;
  if(nz_from_file)
    {
      // Read in n_z
      read_n_of_z(nzfn, dz, n_redshift_bins, nz);
    };
  
  // Check output file
  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };

  // User output
  std::cerr<<"Using cosmology from "<<cosmo_paramfile<<":"<<std::endl;
  std::cerr<<cosmo;
  std::cerr<<"Writing to:"<<outfn<<std::endl;
  
  //Initialize Bispectrum

 
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));

  if(nz_from_file)
    {
      set_cosmology(cosmo, dz, nz_from_file, &nz);
    }
  else
    {
      set_cosmology(cosmo, dz);
    };
  
    
  // Set up thetas for which ApertureStatistics are calculated
  std::vector<double> thetas{0.5, 1, 2, 4, 8, 16, 32}; //Thetas in arcmin
  int N=thetas.size();

  // Borders of integral
  double phiMin=0.0;
  double phiMax=6.28319;
  double lMin=1;

    // Set up cosmologies at which Map^3 is calculated
  // This can probably be done smarter
  // Sets each parameter to N_cosmo values between fac_min*Main Value and fac_max*Main Value
  int N_cosmo=10; //Number of variations for each parameter
  double fac_min=0.9995; //Minimum proportion of main value for each parameter
  double fac_max=1.0005; //Maximum proportion of main value for each parameter
  double fac_bin=(fac_max-fac_min)/N_cosmo;
  
  std::vector<cosmology> cosmos(N_cosmo*7); ///<container for all cosmologies
  for(int i=0; i<N_cosmo; i++)
    {
      double fac=fac_min+i*fac_bin;
      cosmology newCosmo=cosmo;
      newCosmo.h=cosmo.h*fac;
      cosmos.at(i)=newCosmo;

      newCosmo=cosmo;
      newCosmo.sigma8=cosmo.sigma8*fac;
      cosmos.at(i+N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.omb=cosmo.omb*fac;
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.at(i+2*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.ns=cosmo.ns*fac;
      cosmos.at(i+3*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.w=cosmo.w*fac;
      cosmos.at(i+4*N_cosmo)=newCosmo;
      
      newCosmo=cosmo;
      newCosmo.om=cosmo.om*fac;
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.at(i+5*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.ow=cosmo.ow*fac;
      cosmos.at(i+6*N_cosmo)=newCosmo;
    }


  for(int i=0; i<N_cosmo*7; i++)
    {
      std::cout<<"Doing calculations for cosmology "<<i+1<<" of "<<N_cosmo*7<<std::endl;
      auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement
      // Initialize Bispectrum
      if(nz_from_file)
	{
	  set_cosmology(cosmo, dz, nz_from_file, &nz);
	}
      else
	{
	  set_cosmology(cosmo, dz);
	};
      
      
      //Needed for monitoring
      int Ntotal=N;//Total number of bins that need to be calculated
      int step=0;

      out<<cosmos[i].h<<" "<<cosmos[i].sigma8<<" "<<cosmos[i].omb<<" "<<cosmos[i].ns<<" "<<cosmos[i].w<<" "<<cosmos[i].om<<" "<<cosmos[i].ow<<" ";

      
      //Calculate <MapMapMap>(theta1, theta1, theta1)
      // Calculation only for theta1=theta2=theta3
      for (int j=0; j<N; j++)
	{
	  double theta=thetas.at(j)*3.1416/180./60; //Conversion to rad
	  double thetas_calc[3]={theta, theta, theta};
	  //Progress for the impatient user (Thetas in arcmin)
	  step+=1;
	  std::cout<<step<<"/"<<Ntotal<<": Thetas:"<<thetas.at(j)<<" "<<thetas.at(j)<<" "<<thetas.at(j)<<" \r"; //\r is so that only one line is shown
	  std::cout.flush();

	  double Map3=MapMapMap(thetas_calc, phiMin, phiMax, lMin); //Do calculation
	  out<<Map3<<" ";
	  
	};
      out<<std::endl;
      // Stop measuring time and calculate the elapsed time
      auto end = std::chrono::high_resolution_clock::now();
      auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
      
      std::cout<<"Time needed for last cosmology:"<<elapsed.count()*1e-9<<std::endl;
    };
}
