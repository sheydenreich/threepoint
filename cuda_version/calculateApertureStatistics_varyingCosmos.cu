#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cuda_helpers.cuh"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono> //For time measurements
/**
 * @file calculateApertureStatistics_varyingCosmos.cu
 * This executable calculates <MapMapMap> for variations of
 * the cosmological parameters \f$h$\f, \f$\sigma_8$\f, \f$\Omega_b$\f, 
 * \f$n_s$\f, \f$w$\f, \f$\Omega_m$\f, and \f$\Omega_\Lambda$\f 
 * for predefined thetas from the Takahashi+ Bispectrum
 * Code uses CUDA and cubature library  (See https://github.com/stevengj/cubature for documentation)
 * @author Laila Linke
 * @warning Currently only equilateral triangles
 * @warning thetas currently hardcoded
 * @warning Main cosmology is hardcoded (Set to either MS)
 * @todo Thetas should be read from command line
 * @todo cosmology should be read from command line
 */

int main()
{
    

  std::string cosmo_paramfile, outfn;

  if(slics)
    {
      // Set Up Cosmology
      cosmo_paramfile="SLICS_cosmo.dat";
      // Set output file
      outfn="../../results_SLICS/MapMapMap_varyingCosmos.dat";
    }
  else
    {
      // Set Up Cosmology
      cosmo_paramfile="MR_cosmo.dat";
      // Set output file
      outfn="../../results_MR/MapMapMap_varyingCosmos.dat";
    };
  
  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  // Check output file
  std::ofstream out;
  out.open(outfn.c_str());
  if(!out.is_open())
    {
      std::cerr<<"Couldn't open "<<outfn<<std::endl;
      exit(1);
    };

  // User output
  std::cerr<<"Using cosmology:"<<std::endl;
  std::cerr<<cosmo;
  std::cerr<<"Writing to:"<<outfn<<std::endl;
  
  //Initialize Bispectrum

  double dz = cosmo.zmax/((double) n_redshift_bins); //redshift binsize
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96),&A96,48*sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96),&W96,48*sizeof(double)));


    
  // Set up thetas for which ApertureStatistics are calculated
  std::vector<double> thetas{0.5};//{0.5, 1, 2};//, 4, 8, 16, 32}; //Thetas in arcmin
  int N=thetas.size();

  // Borders of integral
  double phiMin=0.0;
  double phiMax=6.28319;
  double lMin=1;

    // Set up cosmologies at which Map^3 is calculated
  // This can probably be done smarter
  // Sets each parameter to N_cosmo values between fac_min*Main Value and fac_max*Main Value
  int N_cosmo=10; //Number of variations for each parameter
  double fac_min=0.9995; //Minimum proportion of main value for each parameter
  double fac_max=1.0005; //Maximum proportion of main value for each parameter
  double fac_bin=(fac_max-fac_min)/N_cosmo;
  
  std::vector<cosmology> cosmos(N_cosmo*7); ///<container for all cosmologies
  for(int i=0; i<N_cosmo; i++)
    {
      double fac=fac_min+i*fac_bin;
      cosmology newCosmo=cosmo;
      newCosmo.h=cosmo.h*fac;
      cosmos.at(i)=newCosmo;

      newCosmo=cosmo;
      newCosmo.sigma8=cosmo.sigma8*fac;
      cosmos.at(i+N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.omb=cosmo.omb*fac;
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.at(i+2*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.ns=cosmo.ns*fac;
      cosmos.at(i+3*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.w=cosmo.w*fac;
      cosmos.at(i+4*N_cosmo)=newCosmo;
      
      newCosmo=cosmo;
      newCosmo.om=cosmo.om*fac;
      newCosmo.omc=newCosmo.om-newCosmo.omb;
      cosmos.at(i+5*N_cosmo)=newCosmo;

      newCosmo=cosmo;
      newCosmo.ow=cosmo.ow*fac;
      cosmos.at(i+6*N_cosmo)=newCosmo;
    }


  for(int i=0; i<N_cosmo*7; i++)
    {
      std::cout<<"Doing calculations for cosmology "<<i+1<<" of "<<N_cosmo*7<<std::endl;
      auto begin=std::chrono::high_resolution_clock::now(); //Begin time measurement
      // Initialize Bispectrum
      set_cosmology(cosmos[i], dz);
     
      
      //Needed for monitoring
      int Ntotal=N;//Total number of bins that need to be calculated
      int step=0;

      out<<cosmos[i].h<<" "<<cosmos[i].sigma8<<" "<<cosmos[i].omb<<" "<<cosmos[i].ns<<" "<<cosmos[i].w<<" "<<cosmos[i].om<<" "<<cosmos[i].ow<<" ";

      
      //Calculate <MapMapMap>(theta1, theta1, theta1)
      // Calculation only for theta1=theta2=theta3
      for (int j=0; j<N; j++)
	{
	  double theta=thetas.at(j)*3.1416/180./60; //Conversion to rad
	  double thetas_calc[3]={theta, theta, theta};
	  //Progress for the impatient user (Thetas in arcmin)
	  step+=1;
	  std::cout<<step<<"/"<<Ntotal<<": Thetas:"<<thetas.at(j)<<" "<<thetas.at(j)<<" "<<thetas.at(j)<<" \r"; //\r is so that only one line is shown
	  std::cout.flush();

	  double Map3=MapMapMap(thetas_calc, phiMin, phiMax, lMin); //Do calculation
	  out<<Map3<<" ";
	  
	};
      out<<std::endl;
      // Stop measuring time and calculate the elapsed time
      auto end = std::chrono::high_resolution_clock::now();
      auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
      
      std::cout<<"Time needed for last cosmology:"<<elapsed.count()*1e-9<<std::endl;
    };
}
