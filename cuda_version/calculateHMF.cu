#include "hip/hip_runtime.h"
#include "apertureStatistics.cuh"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"
#include "helpers.cuh"
#include "halomodel.cuh"

#include <fstream>
#include <iostream>
#include <string>
#include <vector>
/**
 * @file calculateHMF.cu
 * This executable gives out the Halo Mass Function used for the Tri- and Pentaspectrum for consistency tests
 * The Halo Mass Function is the Sheth-Tormen (2001) one
 * Cosmology is read from file
 * @author Laila Linke
 */
int main(int argc, char *argv[])
{
  // Read in command line

  const char *message = R"( 
calculateHMF.x : Wrong number of command line parameters (Needed: 4)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculateHMF.x ../necessary_files/MR_cosmo.dat ../../results_MR/HMF.dat ../necessary_files/nz_MR.dat
)";

  if (argc < 3) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, thetasfn, outfn, nzfn;

  cosmo_paramfile = argv[1];
  outfn = argv[2];
  nzfn = argv[3];
  int Ntomo=1;

    // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open())
  {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  // Read in n_z
  std::vector<std::vector<double>> nzs;
  std::vector<double> nz;
  read_n_of_z(nzfn, n_redshift_bins, cosmo.zmax, nz);
  nzs.push_back(nz);


  // Initialize Bispectrum

  copyConstants();
 double* dev_g_array, * dev_p_array;
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_g_array, Ntomo * n_redshift_bins * sizeof(double)));
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_p_array, Ntomo * n_redshift_bins * sizeof(double)));
  set_cosmology(cosmo, dev_g_array, dev_p_array, &nzs);


  initHalomodel();

  double mmin = 10;
  double mmax = 16;
  int Nbins = 100;
  double mbin = (mmax - mmin) / Nbins;
  double z = 1;

  for (int i = 0; i < Nbins; i++)
  {
    double m = pow(10, mmin + (i + 0.5) * mbin);
    double u = hmf(m, z);
    out << m << " " << u << std::endl;
  }


  hipFree(dev_g_array);
  hipFree(dev_p_array);


  return 0;
}
