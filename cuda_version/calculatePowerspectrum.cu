#include "hip/hip_runtime.h"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"
#include "helpers.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <math.h>

/**
 * @file calculatePowerspectrum.cu
 * This executable gives out the Limberintegrated Revised Halofit Powerspectrum
 * The cosmology is read from file
 * @warning ellMin and ellMax are hardcoded
 * @author Sven Heydenreich
 */
int main(int argc, char *argv[])
{
  const char *message = R"( 
calculatePowerspectrum.x : Wrong number of command line parameters (Needed: 5)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./calculatePowerspectrum.x ../necessary_files/MR_cosmo.dat ../../results_MR/powerspectrum_MR.dat ../necessary_files/nz_MR.dat
)";

  // Read in command line
  double lk_min = -5;
  double lk_max = 1;
  const int n_k = 100;
  double *k = new double[n_k];
  double *z = new double[n_k];
  for (int i = 0; i < n_k; i++)
  {
    double k_temp = lk_min + (lk_max - lk_min) * (i + 0.5) / n_k;
    k[i] = pow(10, k_temp);
    z[i] = 1.;
  }
  double *value = new double[n_k];

  if (argc < 4) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, thetasfn, outfn, nzfn;

  cosmo_paramfile = argv[1];
  outfn = argv[2];
  nzfn = argv[3];

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);

  sigma = 0;
  n = 1;

  // Read in n_z
  std::vector<double> nz;
  read_n_of_z(nzfn, n_redshift_bins, cosmo.zmax, nz);

  // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open())
  {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };

  // User output
  std::cerr << "Using cosmology from " << cosmo_paramfile << ":" << std::endl;
  std::cerr << cosmo;
  std::cerr << "Writing to:" << outfn << std::endl;

  // Initialize Bispectrum

  copyConstants();

  std::cerr << "Using n(z) from " << nzfn << std::endl;
  set_cosmology(cosmo, &nz);

  for (double ell = 1; ell < 5. * pow(10, 4); ell *= 1.05)
  {
    // double ell = ells[i];
    printf("\b\b\b\b\b\b\b\b\b\b\b\b [%.3e]", ell);
    // Output
    out << ell << " " << Pell(ell) << " " << std::endl;
  }
  out.close();

  return 0;
}
