#include "hip/hip_runtime.h"
#include "bispectrum.cuh"
#include "cosmology.cuh"
#include "cuda_helpers.cuh"
#include "helpers.cuh"

#include <fstream>
#include <iostream>
#include <string>
#include <vector>
/**
 * @file testPowerspectrum.cu
 * This executable calculates the nonlinear power spectrum from the
 * Takahashi+ halofit formula
 * @author Sven Heydenreich
 */
int main(int argc, char *argv[]) {
  // Read in command line
  double lk_min =-5;
  double lk_max = 1;
  const int n_k = 100;
  double* k = new double[n_k];
  double* z = new double[n_k];
  for(int i=0;i<n_k;i++)
  {
    double k_temp = lk_min + (lk_max-lk_min)*(i+0.5)/n_k;
    k[i] = pow(10,k_temp);
    z[i] = 1.;
  }
  double* value = new double[n_k];

  const char *message = R"( 
testPowerspectrum.x : Wrong number of command line parameters (Needed: 5)
Argument 1: Filename for cosmological parameters (ASCII, see necessary_files/MR_cosmo.dat for an example)
Argument 2: Outputfilename, directory needs to exist 
Argument 3: 0: use analytic n(z) (only works for MR and SLICS), or 1: use n(z) from file                  
Argument 4 (optional): Filename for n(z) (ASCII, see necessary_files/nz_MR.dat for an example)

Example:
./testPowerspectrum.x ../necessary_files/MR_cosmo.dat ../../results_MR/powerspectrum_MR.dat 1 ../necessary_files/nz_MR.dat
)";

  if (argc < 4) // Give out error message if too few CLI arguments
  {
    std::cerr << message << std::endl;
    exit(1);
  };

  std::string cosmo_paramfile, thetasfn, outfn, nzfn;
  bool nz_from_file = false;

  cosmo_paramfile = argv[1];
  outfn = argv[2];
  nz_from_file = std::stoi(argv[3]);
  if (nz_from_file) {
    nzfn = argv[4];
  };

  // Read in cosmology
  cosmology cosmo(cosmo_paramfile);
  double dz = cosmo.zmax / ((double)n_redshift_bins - 1); // redshift binsize

  // Read in n_z
  std::vector<double> nz;
  if (nz_from_file) {

    read_n_of_z(nzfn, n_redshift_bins, cosmo.zmax, nz);
  };

  // Check if output file can be opened
  std::ofstream out;
  out.open(outfn.c_str());
  if (!out.is_open()) {
    std::cerr << "Couldn't open " << outfn << std::endl;
    exit(1);
  };

  // User output
  std::cerr << "Using cosmology from " << cosmo_paramfile << ":" << std::endl;
  std::cerr << cosmo;
  std::cerr << "Writing to:" << outfn << std::endl;

  // Initialize Bispectrum

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_A96), &A96, 48 * sizeof(double)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_W96), &W96, 48 * sizeof(double)));

  if (nz_from_file) {
    std::cerr << "Using n(z) from " << nzfn << std::endl;
    set_cosmology(cosmo, &nz);
  } else {
    set_cosmology(cosmo);
  };

  get_P_k_nonlinear(k, z, value, n_k);

  // Output
  for (int i = 0; i < n_k; i++) {
      out << k[i] << " " << z[i] << " " << value[i] << " " << std::endl;
  };

  return 0;
}
